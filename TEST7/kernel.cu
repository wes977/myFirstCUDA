#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <cstdlib>

#include <stdio.h>


#define SIZE	1024
#define passwordSize 4


char testPassword[] = "";

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;

	if (i < n)
		c[i] = a[i] + b[i];
}



int main()
{
	int *a, *b, *c, *d;
	int *d_a, *d_b, *d_c, *d_d;

	char password[] = "West";

	double secondsPassed;

	double durationCPU;
	durationCPU = 0;

	double durationGPU;
	durationGPU = 0;

	clock_t startTime = clock(); //Start timer
	clock_t endTime = clock(); //Start timer
	a = (int *)malloc(SIZE * sizeof(int));
	b = (int *)malloc(SIZE * sizeof(int));
	c = (int *)malloc(SIZE * sizeof(int));
	d = (int *)malloc(SIZE * sizeof(int));

	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int));
	hipMalloc(&d_d, SIZE * sizeof(int));
	//Numbers and all that input them and all that fun stuff 
	for (int i = 0; i <= 9; ++i)
	{
		a[i] = i + 48;
		b[i] = i + 48;
		c[i] = i + 48;
		d[i] = i + 48;
	}

	//upper case and all that  
	for (int i = 10; i <= 35; ++i)
	{
		a[i] = i + 55;
		b[i] = i + 55;
		c[i] = i + 55;
		d[i] = i + 55;
	}

	//Lower case  
	for (int i = 36; i <= 62; ++i)
	{
		a[i] = i + 61;
		b[i] = i + 61;
		c[i] = i + 61;
		d[i] = i + 61;
	}

startTime = clock(); //  Resetting the clock 

	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_d, d, SIZE * sizeof(int), hipMemcpyHostToDevice);
	VectorAdd << < 1, SIZE >> >(d_a, d_b, d_c, SIZE);

	//hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	endTime = clock();			// Getting the end time 
	printf("StartTime \t: %d Milliseconds \n", startTime);
	printf("endTime \t: %d Milliseconds \n", endTime);
	printf("Time taken \t: %d Milliseconds \n", (endTime - startTime));


	for (int i = 0; i < 62; ++i)
	{
		printf("c[%d] = %d\n", i, c[i]);
	}

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	printf("Press enter to end everything!");
	getchar();
	return 0;
}
