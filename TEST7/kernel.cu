
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <stdio.h>

using namespace std;

#define SIZE	1024
#define MAX_PRIME	1024
#define passwordSize 4





__global__ void PasswordCrack(int *a, int n)
{

	int i = blockDim.x * blockIdx.x + threadIdx.x;
//	int j = blockDim.x * blockIdx.x + threadIdx.y;

	if (i > 1)
	{
		if (i < n ) // just checking these numbers 
		{

			if (i > 2)
			{
				a[(i) * 2] = 1;

			}
		}
	}


}


int main()
{

	int *a;
	int *d_a;
	int threadsPerBlock = MAX_PRIME;
	int blocksPerGrid = (MAX_PRIME + threadsPerBlock - 1) / threadsPerBlock;



	clock_t startTime = clock(); //Start timer
	clock_t endTime = clock(); //Start timer
	a = (int *)malloc(SIZE * sizeof(int));
	hipMalloc(&d_a, SIZE * sizeof(int));


	for (int j = 1; j <= MAX_PRIME +1; j++)
	{
		a[j] = 0;
	}


	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);


	startTime = clock(); //  Resetting the clock 
	dim3 dimBlock(MAX_PRIME, 1, 1);
	dim3 dimGrid(1);
	PasswordCrack <<<dimGrid, dimBlock >>> (d_a, MAX_PRIME);



	endTime = clock();			// Getting the end time 
	printf("StartTime \t: %d Milliseconds \n", startTime);
	printf("endTime \t: %d Milliseconds \n", endTime);
	printf("Time taken \t: %d Milliseconds \n", (endTime - startTime));
	hipMemcpy(a, d_a, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	int Counter = 0;
	for (int i = 0; i < MAX_PRIME; i++)
	{
		printf("%d = %d\n",i + 1, a[i]);
		if (a[i] == 0)
		{
			Counter++;
		}
	}
	printf("number of primes = %d", Counter);
	//free(a);

	hipFree(d_a);

	printf("Press enter to end everything!");
	getchar();
	return 0;
}
